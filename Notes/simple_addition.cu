#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition on GPU
__global__
void vecAddKernel(float *A_d, float *B_d, float *C_d, int N)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Add if index is within index bounds
    if(i < N) 
        C_d[i] = A_d[i] + B_d[i];
}

// Host function to set up and launch the CUDA kernel
__host__
void vecAdd(float *A_d, float *B_d, float *C_d, int N)
{
    // Grid dimension
    dim3 DimGrid(ceil(N/256.0), 1, 1);

    // Block dimension
    dim3 DimBlock(256, 1, 1);

    // Launch the kernel
    vecAddKernel<<<DimGrid, DimBlock>>>(A_d, B_d, C_d, N);
} 

int main()
{
    // Number of elements in a vector
    int N = 1000;
    
    // Host vectors
    float *A_h, *B_h, *C_h;
    
    // Device vectors
    float *A_d, *B_d, *C_d;
    
    // Size in bytes for N elements
    size_t size = N * sizeof(float);
    
    // Allocate memory on host
    A_h = (float*)malloc(size);
    B_h = (float*)malloc(size);
    C_h = (float*)malloc(size);
    
    // Initialize host vectors
    for(int i = 0; i < N; i++) {
        A_h[i] = i;
        B_h[i] = i * 2;
    }
    
    // Allocate memory on device
    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);
    
    // Copy data from host to device
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);
    
    // Perform vector addition
    vecAdd(A_d, B_d, C_d, N);
    
    // Copy result from device to host
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);
    
    // Print the result
    for(int i = 0; i < 10; i++) {
        printf("%f + %f = %f\n", A_h[i], B_h[i], C_h[i]);
    }
    
    // Free memory
    free(A_h);
    free(B_h);
    free(C_h);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    
    return 0;
}